#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CUDA(fn) do {                           \
        hipError_t err = fn;                   \
        if (err == hipSuccess) break;          \
        printf("CUDA Error: %d\n", err);        \
        return -(int)err;                       \
    } while (0)


#define __NAIVE__ 1
#define __USING_SMEM__ 0
#define __DEBUG__ 1
#define __PROFILE__ 1
#define BLOCK_SIZE 1024

__global__ static void
filter_device_kernel(const float *input, int ilen,
                     const float *kernel, int klen,
                     float *output)
{
    float sum = 0;
    // TODO: WRITE FILTER CODE HERE

    int w = klen / 2;

#if __NAIVE__
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Checking for edge conditions
    int start = tid < w ? 0 : (tid - w);
    int end = tid >= (ilen - w) ? ilen /*(ilen - 1)*/ : (tid + w); //Should use ilen because the loop upper bound below is <

    // Perform the filter operation
    for (int i = start; i < end; i++) {
        sum += input[i] * kernel[i - tid + w];
    }

    if (tid < ilen) output[tid] = sum;
#endif

#if __USING_SMEM__
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int bid = blockIdx.x;
    int local_tid = threadIdx.x;
    int nb = (ilen+BLOCK_SIZE-1)/BLOCK_SIZE;
 
    //int size = BLOCK_SIZE+2*w;
    extern __shared__ float input_shared[]; 

    if(local_tid>0 && local_tid<BLOCK_SIZE-1)
      input_shared[local_tid+w] = input[tid];
    else if (local_tid == 0)
      if (bid>0)
        for (unsigned i=0; i<=w; i++)
          input_shared[w-i] = input[tid-i];
      else //bid == 0
        for (unsigned i=0; i<=w; i++)
          input_shared[w-i] = input[tid];
    else // local_tid == BLOCK_SIZE-1
      if(bid<nb-1)
        for (unsigned i=0; i<=w; i++)
          input_shared[local_tid+w+i] = input[tid+i];
      else //bid == nb-1
        for (unsigned i=0; i<=w; i++)
          input_shared[local_tid+w+i] = input[tid];

    __syncthreads();

    int start = tid  < w ? 0  : local_tid - w;
    int end = tid >= (ilen - w) ? BLOCK_SIZE - 1  : local_tid + w;

    for (int i = start; i < end; i++) {
        sum += input_shared[i] * kernel[i - local_tid + w];
    }

    if (tid < ilen) output[tid] = sum;
#endif

}

hipError_t filter_device(const float *input , int ilen,
                          const float *kernel, int klen,
                          float *output)
{
    dim3 blocks(1,1), threads(1,1);
    // TODO: CALCULATE OPTIMUM BLOCKS AND THREADS HERE
#if __NAIVE__
    blocks.x = ilen/klen; blocks.y = 1;
    threads.x = klen; threads.y = 1;

    filter_device_kernel<<<blocks, threads>>>(input, ilen,
                                              kernel, klen,
                                              output);
#endif

#if __USING_SMEM__
    blocks.x = ilen+(BLOCK_SIZE - 1)/BLOCK_SIZE; blocks.y = 1;
    threads.x = BLOCK_SIZE; threads.y = 1;

    filter_device_kernel<<<blocks, threads, (BLOCK_SIZE+klen)*sizeof(float)>>>(input, ilen,
                                              kernel, klen,
                                              output);
#endif 

    return hipSuccess;
}


hipError_t filter_host(const float *input , int ilen,
                        const float *kernel, int klen,
                        float *output)
{
    int w = klen / 2;

    for (int idx = 0; idx < ilen; idx++) {

        // Checking for edge conditions
        int start = idx < w ? 0 : (idx - w);
        int end = idx >= (ilen - w) ? (ilen - 1) : (idx + w);

        // Perform the filter operation
        float sum = 0;
        for (int i = start; i < end; i++) {
            sum += input[i] * kernel[i - idx + w];
        }

        output[idx] = sum;
    }

    return hipSuccess;
}


#define ILEN 1024 * 1024
#define KLEN 32

int main()
{
    float *h_input, *h_kernel;
    float *h_output, *h_result;

    float *d_input, *d_kernel;
    float *d_output;

    size_t ibytes = ILEN * sizeof(float);
    size_t kbytes = KLEN * sizeof(float);

    // Allocate memory
    h_input  = (float *)malloc(ibytes);
    h_kernel = (float *)malloc(kbytes);
    h_output = (float *)malloc(ibytes);
    h_result = (float *)malloc(ibytes);

    CUDA(hipMalloc(&d_input , ibytes));
    CUDA(hipMalloc(&d_kernel, kbytes));
    CUDA(hipMalloc(&d_output, ibytes));

    // Generate data
    for (int i = 0; i < ILEN; i++) h_input[i]  = (float)(rand() % 100) / 100.0;
    for (int i = 0; i < KLEN; i++) h_kernel[i] = (float)(rand() % 100) / 100.0;

    // Send data to GPU
    CUDA(hipMemcpy(d_input, h_input, ibytes, hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_kernel, h_kernel, kbytes, hipMemcpyHostToDevice));

#if __PROFILE__
    timeval start, stop, diff;

    // Calculate
    gettimeofday(&start, NULL);
#endif
    CUDA(filter_host(h_input, ILEN, h_kernel, KLEN, h_output));
#if __PROFILE__
    gettimeofday(&stop, NULL);
    timersub(&stop, &start, &diff);
    printf("Time on CPU : %ld us\n", diff.tv_sec * 1000000 + diff.tv_usec);

    gettimeofday(&start, NULL);
#endif
    CUDA(filter_device(d_input, ILEN, d_kernel, KLEN, d_output));
#if __PROFILE__
    gettimeofday(&stop, NULL);
    timersub(&stop, &start, &diff);
    printf("Time on GPU : %ld us\n", diff.tv_sec * 1000000 + diff.tv_usec);
#endif

    // Copy data back from GPU
    CUDA(hipMemcpy(h_result, d_output, ibytes, hipMemcpyDeviceToHost));

    // Error checking
    float err = 0.0;
    for (int i = 0; i < ILEN; i++) {
#if __DEBUG__
        if(i < 10)
	  printf("h_result[%d]=%lf, h_output[%d]=%lf\n", i, h_result[i], i, h_output[i]);
#endif
        float diff = h_result[i] - h_output[i];
        err = err + diff * diff;
    }
    err = err / ILEN;
    printf("Error: %lf\n", err);

    // TODO: BENCHMARK filter_host and filter_device

    free(h_input);
    free(h_kernel);
    free(h_output);
    free(h_result);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    return 0;
}
