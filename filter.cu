#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CUDA(fn) do {                           \
        hipError_t err = fn;                   \
        if (err == hipSuccess) break;          \
        printf("CUDA Error: %d\n", err);        \
        return -(int)err;                       \
    } while (0)

__global__ static void
filter_device_kernel(const float *input, int ilen,
                     const float *kernel, int klen,
                     float *output)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0;
    // TODO: WRITE FILTER CODE HERE
    int w = klen / 2;

    // Checking for edge conditions
    int start = tid < w ? 0 : (tid - w);
    int end = tid >= (ilen - w) ? ilen /*(ilen - 1)*/ : (tid + w); //Should use ilen because the loop upper bound below is <

    // Perform the filter operation
    for (int i = start; i < end; i++) {
        sum += input[i] * kernel[i - tid + w];
    }

    if (tid < ilen) output[tid] = sum;
}

hipError_t filter_device(const float *input , int ilen,
                          const float *kernel, int klen,
                          float *output)
{
    dim3 blocks(1,1), threads(1,1);
    // TODO: CALCULATE OPTIMUM BLOCKS AND THREADS HERE
    blocks.x = ilen/klen; blocks.y = 1;
    threads.x = klen; threads.y = 1; 

    filter_device_kernel<<<blocks, threads>>>(input, ilen,
                                              kernel, klen,
                                              output);

    return hipSuccess;
}


hipError_t filter_host(const float *input , int ilen,
                        const float *kernel, int klen,
                        float *output)
{
    int w = klen / 2;

    for (int idx = 0; idx < ilen; idx++) {

        // Checking for edge conditions
        int start = idx < w ? 0 : (idx - w);
        int end = idx >= (ilen - w) ? (ilen - 1) : (idx + w);

        // Perform the filter operation
        float sum = 0;
        for (int i = start; i < end; i++) {
            sum += input[i] * kernel[i - idx + w];
        }

        output[idx] = sum;
    }

    return hipSuccess;
}


#define ILEN 1024 * 1024
#define KLEN 32

int main()
{
    float *h_input, *h_kernel;
    float *h_output, *h_result;

    float *d_input, *d_kernel;
    float *d_output;

    size_t ibytes = ILEN * sizeof(float);
    size_t kbytes = KLEN * sizeof(float);

    // Allocate memory
    h_input  = (float *)malloc(ibytes);
    h_kernel = (float *)malloc(kbytes);
    h_output = (float *)malloc(ibytes);
    h_result = (float *)malloc(ibytes);

    CUDA(hipMalloc(&d_input , ibytes));
    CUDA(hipMalloc(&d_kernel, kbytes));
    CUDA(hipMalloc(&d_output, ibytes));

    // Generate data
    for (int i = 0; i < ILEN; i++) h_input[i]  = (float)(rand() % 100) / 100.0;
    for (int i = 0; i < KLEN; i++) h_kernel[i] = (float)(rand() % 100) / 100.0;

    // Send data to GPU
    CUDA(hipMemcpy(d_input, h_input, ibytes, hipMemcpyHostToDevice));
    CUDA(hipMemcpy(d_kernel, h_kernel, kbytes, hipMemcpyHostToDevice));

    timeval start, stop, diff;

    // Calculate
    gettimeofday(&start, NULL);
    CUDA(filter_host(h_input, ILEN, h_kernel, KLEN, h_output));
    gettimeofday(&stop, NULL);
    timersub(&stop, &start, &diff);
    printf("Time on CPU : %ld us\n", diff.tv_sec * 1000000 + diff.tv_usec);

    gettimeofday(&start, NULL);
    CUDA(filter_device(d_input, ILEN, d_kernel, KLEN, d_output));
    gettimeofday(&stop, NULL);
    timersub(&stop, &start, &diff);
    printf("Time on GPU : %ld us\n", diff.tv_sec * 1000000 + diff.tv_usec);

    // Copy data back from GPU
    CUDA(hipMemcpy(h_result, d_output, ibytes, hipMemcpyDeviceToHost));

    // Error checking
    float err = 0.0;
    for (int i = 0; i < ILEN; i++) {
        //if(i < 10)
	  //printf("h_result[%d]=%lf, h_output[%d]=%lf\n", i, h_result[i], i, h_output[i]);
        float diff = h_result[i] - h_output[i];
        err = err + diff * diff;
    }
    err = err / ILEN;
    printf("Error: %lf\n", err);

    // TODO: BENCHMARK filter_host and filter_device

    free(h_input);
    free(h_kernel);
    free(h_output);
    free(h_result);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    return 0;
}
